#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>

__global__ void custom_kernel(float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = input[idx] * 2.0f;  // Example operation
    }
}

torch::Tensor custom_op(torch::Tensor input) {
    auto output = torch::zeros_like(input);
    int size = input.numel();
    int threads = 1024;
    int blocks = (size + threads - 1) / threads;

    custom_kernel<<<blocks, threads>>>(input.data_ptr<float>(), output.data_ptr<float>(), size);
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("custom_op", &custom_op, "Custom CUDA kernel");
}